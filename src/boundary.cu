#include "hip/hip_runtime.h"
#include "boundary.h"
#include "boundary_kernels.cuh"

BoundaryCondition::BoundaryCondition(BoundaryType type, BoundaryDirection dimension, std::function<float(float,float,float)> value)
    :_type(type),_dimension(dimension){
        _single_value=0;
        _value_fn=value;
}
     // 3d constructor
BoundaryCondition::BoundaryCondition(BoundaryType type, BoundaryDirection dimension, float value)
    :_type(type),_dimension(dimension),_value(value){
        _single_value=1;
    } //2d constructor
void BoundaryCondition::initalize(field * myField){
    // ok first thing we need to do is grab the pointer to the field
    // _field = myEvolver->fieldsMap[_fieldName]; // TO DO Name validation
    _fieldSize = myField->get_size();
    _fieldSpacing = myField->get_spacing();
    _with_cuda = myField->isCUDA;
    _boundarySize = _fieldSize;
    _boundarySize[_dimension/2] = 1; // we don't need to iterate over the demension we're setting the boundary on

    // now that we have that we have dimensional information, we need to check to see if we need to calculate values for the boundary (if its not a single value)
    if (!_single_value) {
        // if we have to do that allocate the space here, figure out the values and then transfer them to the GPU if we're using it
        // the numebr of values we needs is the product of the dimensions that aren't this one
        
        long boundarySize = 1;
        for (long i = 0; i<3; i++){
           boundarySize*=_boundarySize[i];
        }
        _values = new float[boundarySize];
        long index = 0;
        std::array<float,3> position;
        for (int iz = 0; iz<_boundarySize[2]; iz++){
            for (int iy = 0; iy<_boundarySize[1]; iy++){
                for (int ix = 0; ix<_boundarySize[0]; ix++){
                    position = {ix*_fieldSpacing[0],iy*_fieldSpacing[1],iz*_fieldSpacing[2]};
                    // if we're on the right boundary we need to correct the value of that position
                    if (_dimension%2 == 1) {
                        // right boundary 
                        position[_dimension/2]=(_fieldSize[_dimension/2]-1)*_fieldSpacing[_dimension/2];// hmmm might need to think carefully about how to encorpurate boundary layers here. for now ignore
                    }
                    _values[index] = _value_fn(position[0],position[1],position[2]);
                    index++;
                }
            }
        }
        if (_with_cuda){
            hipMalloc(reinterpret_cast<void **>(&d_values), boundarySize * sizeof(float));
            hipMemcpy(d_values, _values, boundarySize * sizeof(float), hipMemcpyHostToDevice);
        }
    }
    
    if (_with_cuda){
        _threadDim=dim3(32,32,32);
        switch (_dimension/2){
            case 0:
                _threadDim.x = 1;
                break;
            case 1:
                _threadDim.y = 1;
                break;
            case 2:
                _threadDim.z = 1;
                break;
        }
        // for 128x128 in, boundary in the x dim we have
        int bx = (_boundarySize[0]+_threadDim.x-1)/_threadDim.x; // 1+1-1/1 = 1
        int by = (_boundarySize[1]+_threadDim.y-1)/_threadDim.y; // (128 + 32 -1)/32 =  
        int bz = (_boundarySize[2]+_threadDim.z-1)/_threadDim.z;
        _blockDim = dim3(bx,by,bz);
    }
}
long BoundaryCondition::flatten_index(std::array<int,3> dimension_index)
{
    // index = xi + nx *yi +nx *ny *zi 
    return dimension_index[0]+dimension_index[1]*_fieldSize[0]+dimension_index[2]*_fieldSize[0]*_fieldSize[1];
}
void BoundaryCondition::operator()(float2* fieldValues)
{
    switch (_type){
        case BoundaryType::Dirichlet:
            applyDirichlet(fieldValues);
            break;
        case BoundaryType::VonNeumann:
            applyVonNeumann(fieldValues);
            break;

    }
} 
void BoundaryCondition::applyDirichlet(float2* fieldValues) 
{
    if (_with_cuda)
    {
        bool leftwall = !(_dimension%2);
        dim3 field_size = dim3(_fieldSize[0], _fieldSize[1], _fieldSize[2]);
        dim3 boundary_size = dim3(_boundarySize[0], _boundarySize[1], _boundarySize[2]);

        if (_single_value) 
        {
            applyDiricheltSingleValue_gpu(fieldValues,_value,_depth,_dimension/2, leftwall, field_size,  boundary_size,  _blockDim,  _threadDim);
        }
        else {
            applyDiricheltMultipleValue_gpu(fieldValues,d_values,_depth,_dimension/2, leftwall, field_size,  boundary_size,  _blockDim,  _threadDim);
        }

    } 
    
    else 
    {
        long valueIndex = 0;
        long fieldIndex = 0;
        std::array<int,3> dimension_index;
        for (int iz = 0; iz<_boundarySize[2]; iz++) {
            for (int iy = 0; iy<_boundarySize[1]; iy++) {
                for (int ix = 0; ix<_boundarySize[0]; ix++) {
                    for (int ib = 0; ib < _depth; ib ++) {
                        dimension_index = {ix,iy,iz};
                        if (_dimension%2 == 0) {
                            // left wall
                            dimension_index[_dimension/2]=ib;
                        }
                        if (_dimension%2 == 1){
                            // right wall
                            dimension_index[_dimension/2]=_fieldSize[_dimension/2]-ib-1;
                        }

                        fieldIndex = flatten_index(dimension_index);
                        if (_single_value){// add aliased array
                            fieldValues[fieldIndex].x=_value;
                        } else {
                            fieldValues[fieldIndex].x=_values[valueIndex];
                        }

                    }
                    valueIndex++;
                }
            }
        }
    }
}
void BoundaryCondition::applyVonNeumann(float2* fieldValues){
    if (_with_cuda)
    {
        bool leftwall = !(_dimension%2);
        dim3 field_size = dim3(_fieldSize[0], _fieldSize[1], _fieldSize[2]);
        dim3 boundary_size = dim3(_boundarySize[0], _boundarySize[1], _boundarySize[2]);
        float h = _fieldSpacing[_dimension/2];
        if (_single_value) 
        {
            applyVonNuemannSingleValue_gpu(fieldValues,_value,_depth,_dimension/2, leftwall, field_size,  boundary_size,h,  _blockDim,  _threadDim);
        }
        else {
            applyVonNuemannMultipleValue_gpu(fieldValues,d_values,_depth,_dimension/2, leftwall, field_size,  boundary_size, h ,  _blockDim,  _threadDim);
        }

    } else {
        long valueIndex = 0;
        long fieldIndex = 0;
        long fieldIndexOneIn = 0;
        std::array<int,3> dimension_index;
        std::array<int,3> dimension_index_one_in;
        for (int iz = 0; iz<_boundarySize[2]; iz++) {
            for (int iy = 0; iy<_boundarySize[1]; iy++) {
                for (int ix = 0; ix<_boundarySize[0]; ix++) {
                    for (int ib = 0; ib < _depth; ib ++) {
                        dimension_index = {ix,iy,iz};
                        dimension_index_one_in=dimension_index;
                        // for von nueman we have to start from the inside and go out to keep the deriviate correct
                        if (_dimension%2 == 0) {
                            // left wall
                            dimension_index[_dimension/2]=(_depth-ib-1);
                            dimension_index_one_in[_dimension/2]=(_depth-ib);

                        }
                        if (_dimension%2 == 1){
                            // right wall
                            dimension_index[_dimension/2]=_fieldSize[_dimension/2]-1-(ib-_depth-1);
                            dimension_index_one_in[_dimension/2]=_fieldSize[_dimension/2]-1-(ib-_depth);

                        }

                        fieldIndex = flatten_index(dimension_index);
                        fieldIndexOneIn = flatten_index(dimension_index_one_in);
                        // (x[one in]-x)/dx = value
                        // x = x[one in] - dx *value;
                        if (_single_value){
                            fieldValues[fieldIndex].x=fieldValues[fieldIndexOneIn].x-_fieldSpacing[_dimension/2]*_value;
                        } else {
                            fieldValues[fieldIndex].x=fieldValues[fieldIndexOneIn].x-_fieldSpacing[_dimension/2]*_values[valueIndex];
                        }

                    }
                    valueIndex++;
                }
            }
        }
    }
}
